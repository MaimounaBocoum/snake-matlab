#include "hip/hip_runtime.h"
__constant__ float m_speedOfSound;
__constant__ float m_invSpeedOfSound;
__constant__ float m_piezoPitch;
__constant__ float m_sampFreq;
__constant__ float m_rOrigin;
__constant__ float m_fNumber;
__constant__ float m_peakDelay;
__constant__ float m_xSource[NB_MAX_SOURCES];
__constant__ float m_zSource[NB_MAX_SOURCES];
__constant__ float m_zApex;
__constant__ float m_linePitch;
__constant__ float m_pixelPitch;
__constant__ float m_lambda;
__constant__ float m_thetaOrigin;
__constant__ int   m_nbPiezos;
__constant__ int   m_nbSources;
__constant__ int   m_channelOffset;
__constant__ int   m_firstSample;
__constant__ int   m_nbSamples;
__constant__ int   m_nbLinesPerRecon;
__constant__ int   m_nbPixelsPerLine;
__constant__ int   m_nbRecon;
__constant__ int   m_normMode;
__constant__ int   m_synthAcq;
__constant__ int   m_frame_per_frame;
__constant__ int   m_idxTransmitToBeamform;
__constant__ int   m_idxFrame;
__constant__ int   m_nbImages;
__constant__ int   m_usegpu;

__global__ void Mafalda_rf_beamform_kernel_rectgrid200_source_frames(const short* pInRF,const int bufferSize,float* pOutImageRF, int frame)
{    
// output pixel coords
    int ix = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;  // recon
    int iy = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;  // sample (depth)

    if(ix< m_nbRecon && iy < m_nbPixelsPerLine)
     float thetaRecon = ix*m_nbLinesPerRecon*m_linePitch + m_thetaOrigin;
   
		float RfromApex = (float)iy * m_pixelPitch + m_rOrigin;
        float aperture = RfromApex/m_fNumber;
		int halfAperture = (int) roundf(0.5f * aperture /m_piezoPitch);
        int firstChannel0 = m_nbPiezos/2 - halfAperture; 
        int lastChannel0 = m_nbPiezos/2 + halfAperture;
        
		const float w0 = -1.0f/6.0f;
		const float w1 = 0.5f;
		const float w2 = -0.5f;
		const float w3 = 1.0f/6.0f;
		const float eps = 0.000001f;
		const float normal_fudge = 1.0f;

		int firstChannel = 0;
		int lastChannel  = min(NB_ACQ_CHANNELS,m_nbPiezos);		

	    float normCoef = eps;
		float RF;
		int offset;
        
		for (int channel = firstChannel; channel < lastChannel; channel++)
		{
		    offset = (channel % NB_ACQ_CHANNELS) * m_channelOffset + m_synthAcq * (channel / NB_ACQ_CHANNELS) * m_nbSamples ;
            offset+= (1+m_synthAcq)*m_idxFrame*m_nbSamples ;
            
		    float apodCoef = 1;//%apod(channel - firstChannel0, lastChannel0 - firstChannel0+1);
		    float xPiezo = (float)(channel + 0.5f - m_nbPiezos/2) * m_piezoPitch;
		    float zPiezo = m_zApex;

		        const short* channelStartPtr = pInRF + offset;// + (1+m_synthAcq)*m_nbSamples;
		        		        
		        for (int iPixel = 0; iPixel < m_nbLinesPerRecon; iPixel++)
		        {
                    int columnOffset = (ix*m_nbLinesPerRecon+iPixel) * m_nbPixelsPerLine;
                    float thetaLine = thetaRecon + iPixel * m_linePitch;
                    
                    float X = RfromApex*sinf(thetaLine);
                    float Z = RfromApex*cosf(thetaLine);
                    float forwardDelay = sqrtf((X-m_xSource[frame])*(X-m_xSource[frame]) + (Z-m_zSource[frame])*(Z-m_zSource[frame]))*m_invSpeedOfSound;
                    float returnDelay  = sqrtf((Z-zPiezo) * (Z-zPiezo) + (X - xPiezo) * (X - xPiezo)) * m_invSpeedOfSound;
		            float timeDelay = forwardDelay + returnDelay+(float)(2e-6); //+ m_timeOrigin[iSource];
                   
		            int delay = (int) floorf(timeDelay * m_sampFreq) - m_firstSample;
                    
					float deltaDelay = timeDelay * m_sampFreq - m_firstSample - float(delay); 
				   

				    if(delay < 0) 
                    {
                        delay = 1 ;
						deltaDelay = 0.0;
                    }
                    
                    if( 2*delay-2+1 + offset >= bufferSize ) 
                    {
                        delay = 1 ;
						deltaDelay = 0.0;
                    }

		          RF = (1-deltaDelay)*channelStartPtr[delay] + (deltaDelay)*channelStartPtr[delay+1];

                  pOutImageRF[columnOffset + iy] += RF*apodCoef;
		        }

		    normCoef += apodCoef * apodCoef;
		}

		if (m_normMode < 100)
		{
		    for (int iPixel = 0; iPixel < m_nbLinesPerRecon; iPixel++)
		    {
                int columnOffset = (ix*m_nbLinesPerRecon+iPixel) * m_nbPixelsPerLine;
                pOutImageRF[columnOffset + iy] /= normCoef;
		        pOutImageRF[columnOffset + iy] *= normal_fudge;
		    }
		}
		else
		{
		    for (int iPixel = 0; iPixel < m_nbLinesPerRecon; iPixel++)
		    {
                int columnOffset = (ix*m_nbLinesPerRecon+iPixel) * m_nbPixelsPerLine;
                pOutImageRF[columnOffset + iy] /= sqrtf(normCoef);
		    }
		}
    }
}
